#include "hip/hip_runtime.h"
// -*- mode: c; -*-

#include <stdio.h>
#include <jetrec/hip/hip_runtime.h>

#define CUDA_CHECK_SUCESS(result) \
	if(result != hipSuccess) { \
        fprintf(stderr, "%s:%d: CUDA driver error %x\n", \
				__FILE__, __LINE__, result); \
		switch(result) { \
		case hipErrorInvalidValue: \
			fprintf(stderr, " (%s)", "hipErrorInvalidValue"); \
			break; \
		case hipErrorOutOfMemory: \
			fprintf(stderr, " (%s)", "hipErrorOutOfMemory"); \
			break; \
		case hipErrorNotInitialized: \
			fprintf(stderr, " (%s)", "hipErrorNotInitialized"); \
			break; \
		case hipErrorNoDevice: \
			fprintf(stderr, " (%s)", "hipErrorNoDevice"); \
			break; \
		case hipErrorInvalidDevice: \
			fprintf(stderr, " (%s)", "hipErrorInvalidDevice"); \
			break; \
		case hipErrorInvalidImage: \
			fprintf(stderr, " (%s)", "hipErrorInvalidImage"); \
			break; \
		case hipErrorInvalidContext: \
			fprintf(stderr, " (%s)", "hipErrorInvalidContext"); \
			break; \
		case hipErrorContextAlreadyCurrent: \
			fprintf(stderr, " (%s)", \
					"hipErrorContextAlreadyCurrent"); \
			break; \
		case hipErrorMapFailed: \
			fprintf(stderr, " (%s)", "hipErrorMapFailed"); \
			break; \
		case hipErrorUnmapFailed: \
			fprintf(stderr, " (%s)", "hipErrorUnmapFailed"); \
			break; \
		case hipErrorArrayIsMapped: \
			fprintf(stderr, " (%s)", "hipErrorArrayIsMapped"); \
			break; \
		case hipErrorAlreadyMapped: \
			fprintf(stderr, " (%s)", "hipErrorAlreadyMapped"); \
			break; \
		case hipErrorNoBinaryForGpu: \
			fprintf(stderr, " (%s)", \
					"hipErrorNoBinaryForGpu"); \
			break; \
		case hipErrorAlreadyAcquired: \
			fprintf(stderr, " (%s)", \
					"hipErrorAlreadyAcquired"); \
			break; \
		case hipErrorNotMapped: \
			fprintf(stderr, " (%s)", "hipErrorNotMapped"); \
			break; \
		case hipErrorInvalidSource: \
			fprintf(stderr, " (%s)", "hipErrorInvalidSource"); \
			break; \
		case hipErrorFileNotFound: \
			fprintf(stderr, " (%s)", "hipErrorFileNotFound"); \
			break; \
		case hipErrorInvalidHandle: \
			fprintf(stderr, " (%s)", "hipErrorInvalidHandle"); \
			break; \
		case hipErrorNotFound: \
			fprintf(stderr, " (%s)", "hipErrorNotFound"); \
			break; \
		case hipErrorLaunchFailure: \
			fprintf(stderr, " (%s)", "hipErrorLaunchFailure"); \
			break; \
		case hipErrorLaunchOutOfResources: \
			fprintf(stderr, " (%s)", \
					"hipErrorLaunchOutOfResources"); \
			break; \
		case hipErrorLaunchTimeOut: \
			fprintf(stderr, " (%s)", "hipErrorLaunchTimeOut"); \
			break; \
		case CUDA_ERROR_LAUNCH_INCOMPATIBLE_TEXTURING: \
			fprintf(stderr, " (%s)", \
					"CUDA_ERROR_LAUNCH_INCOMPATIBLE_TEXTURING"); \
			break; \
		case hipErrorUnknown: \
			fprintf(stderr, " (%s)", "hipErrorUnknown"); \
			break; \
		}; \
		fprintf(stderr, "\n"); \
        exit(EXIT_FAILURE); \
	}

#define CUFFT_CHECK_SUCESS(result) \
	if(result != HIPFFT_SUCCESS) { \
        fprintf(stderr, "%s:%d: CUFFT driver error %x", \
				__FILE__, __LINE__, result); \
		switch(result) { \
		case HIPFFT_INVALID_PLAN: \
			fprintf(stderr, " (%s)", "HIPFFT_INVALID_PLAN"); \
			break; \
		case HIPFFT_ALLOC_FAILED: \
			fprintf(stderr, " (%s)", "HIPFFT_ALLOC_FAILED"); \
			break; \
		case HIPFFT_INVALID_TYPE: \
			fprintf(stderr, " (%s)", "HIPFFT_INVALID_TYPE"); \
			break; \
		case HIPFFT_INVALID_VALUE: \
			fprintf(stderr, " (%s)", "HIPFFT_INVALID_VALUE"); \
			break; \
		case HIPFFT_INTERNAL_ERROR: \
			fprintf(stderr, " (%s)", "HIPFFT_INTERNAL_ERROR"); \
			break; \
		case HIPFFT_EXEC_FAILED: \
			fprintf(stderr, " (%s)", "HIPFFT_EXEC_FAILED"); \
			break; \
		case HIPFFT_SETUP_FAILED: \
			fprintf(stderr, " (%s)", "HIPFFT_SETUP_FAILED"); \
			break; \
		case HIPFFT_INVALID_SIZE: \
			fprintf(stderr, " (%s)", "HIPFFT_INVALID_SIZE"); \
			break; \
		}; \
		fprintf(stderr, "\n"); \
        exit(EXIT_FAILURE); \
	}

static __device__ __host__ inline float2
complex_scale(const float2 a, const float s)
{
    float2 c = {a.x * s, a.y * s};

    return c;
}

static __device__ __host__ inline float2
complex_multiply(const float2 a, const float2 b)
{
    float2 c = {a.x * b.x - a.y * b.y, a.x * b.y + a.y * b.x};

    return c;
}

static __global__ void
multiply_scale(float2 a[], const float2 b[], int size, float scale)
{
    const int num_threads = blockDim.x * gridDim.x;
    const int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    for(int i = thread_id; i < size; i += num_threads)
        a[i] = complex_scale(complex_multiply(a[i], b[i]), scale);     
}

void cufft_multiply_scale_32_256(float2 a[], const float2 b[],
								 int size, float scale)
{
	multiply_scale<<<32, 256>>>(a, b, size, scale);

	const hipError_t result = hipDeviceSynchronize();

	CUDA_CHECK_SUCESS(result);
}

void cuda_device_init(void)
{
	hipError_t result;
    int device_count;
    int dev;

	result = hipGetDeviceCount(&device_count);
	CUDA_CHECK_SUCESS(result);
    if(device_count == 0) {
        fprintf(stderr, "%s:%d: no device found\n",
				__FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    for(dev = 0; dev < device_count; dev++) {
        hipDeviceProp_t device_prop;

        result = hipGetDeviceProperties(&device_prop, dev);
		CUDA_CHECK_SUCESS(result);
        if(device_prop.major >= 1)
            break;
    }
    if(dev == device_count) {
        fprintf(stderr, "%s:%d: no device supporting CUDA found\n",
				__FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
    else {
        result = hipSetDevice(dev);
		CUDA_CHECK_SUCESS(result);
	}
}

void cuda_malloc(void **device_ptr, size_t size)
{
	const hipError_t result = hipMalloc(device_ptr, size);

	CUDA_CHECK_SUCESS(result);
}

void cuda_memcpy(void *destination, const void *source,
				 size_t count, enum hipMemcpyKind kind)
{
	const hipError_t result =
		hipMemcpy(destination, source, count, kind);

	CUDA_CHECK_SUCESS(result);
}

void cuda_free(void *device_ptr)
{
	const hipError_t result = hipFree(device_ptr);

	CUDA_CHECK_SUCESS(result);
}

void cufft_plan_1d(hipfftHandle *plan, int nx, hipfftType type,
				   int batch)
{
	const hipfftResult result = hipfftPlan1d(plan, nx, type, batch);

	CUFFT_CHECK_SUCESS(result);
}

void cufft_plan_2d(hipfftHandle *plan, int nx, int ny,
				   hipfftType type)
{
	const hipfftResult result = hipfftPlan2d(plan, nx, ny, type);

	CUFFT_CHECK_SUCESS(result);
}

void cufft_plan_3d(hipfftHandle *plan, int nx, int ny, int nz,
				   hipfftType type)
{
	const hipfftResult result = hipfftPlan3d(plan, nx, ny, nz, type);

	CUFFT_CHECK_SUCESS(result);
}

void cufft_destroy(hipfftHandle plan)
{
	const hipfftResult result = hipfftDestroy(plan);

	CUFFT_CHECK_SUCESS(result);
}

void cufft_exec_c2c(hipfftHandle plan, hipfftComplex *input_data,
					hipfftComplex *output_data, int direction)
{
	const hipfftResult result =
		hipfftExecC2C(plan, input_data, output_data, direction);

	CUFFT_CHECK_SUCESS(result);
}

void cufft_exec_r2c(hipfftHandle plan, hipfftReal *input_data,
					hipfftComplex *output_data)
{
	const hipfftResult result =
		hipfftExecR2C(plan, input_data, output_data);

	CUFFT_CHECK_SUCESS(result);
}

void cufft_exec_c2r(hipfftHandle plan, hipfftComplex *input_data,
					hipfftReal *output_data)
{
	const hipfftResult result =
		hipfftExecC2R(plan, input_data, output_data);

	CUFFT_CHECK_SUCESS(result);
}

